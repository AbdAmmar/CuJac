
#include <hip/hip_runtime.h>



__global__ void compute_kernel(int ntx, int nty, int nty_local, int nWorkers, double h, double *u_old, double *u_new) {

    int tid;

    int l, ll;
    int j, jj0, jj1, jj2, jy;

    double x, y, y_tmp;

    const double h_ct = 2.0 * h * h;

    tid = threadIdx.x + blockIdx.x * blockDim.x;


    while (tid < nWorkers) {

        jj0 = nty_local * tid;

        jy = jj0 - 2 * tid - 1;

        for(j = 1; j < nty_local-1; j++) {
        
            y = (double) (jy + j) * h;
            y_tmp = y * (y - 1.0);
        
            jj1 = (jj0 + j) * ntx;
        
            for(l = 1; l < ntx-1; l++) {
        
                x = (double) l * h;
        
                ll = jj1 + l;
        
                u_new[ll] = 0.25 * ( u_old[ll - 1] + u_old[ll + 1] + u_old[ll - ntx] + u_old[ll + ntx] 
                                   - h_ct * (x * (x - 1.0) + y_tmp) ) ;
            }
        }
        
        if(jj0 != 0) {
            jj1 = (jj0 + 1) * ntx;
            jj2 = jj1 - ntx;
            for(l = 0; l < ntx; l++) {
                u_new[jj2 + l] = u_new[jj1 + l];
            }
        } else {
            for(l = 0; l < ntx; l++) {
                u_new[ntx + l] = 0.0;
            }
        }
      
        if(jj0 + nty_local != nty) {
            jj1 = (jj0 + nty_local) * ntx;
            jj2 = jj1 - ntx;
            for(l = 0; l < ntx; l++) {
                u_new[jj1 - l - 1] = u_new[jj2 - l - 1];
            }
        } else {
            jj1 = (nty - 1) * ntx;
            for(l = 0; l < ntx; l++) {
                u_new[jj1 - l - 1] = 0.0;
            }
        }

        tid += blockDim.x * gridDim.x;
    }
}


extern "C" void compute(int nBlocks, int blockSize, int ntx, int nty, int nty_local, int nWorkers, double h, double *u_old, double *u_new) {

    compute_kernel<<<nBlocks, blockSize>>>(ntx, nty, nty_local, nWorkers, h, u_old, u_new);

}




