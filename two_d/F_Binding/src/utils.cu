#include <hip/hip_runtime.h>

#include <stdio.h>



extern "C" void checkCudaErrors(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        printf("CUDA Error: %f - %f", msg, hipGetErrorString(err));
        exit(0);
    }
}



__global__ void max_error_kernel(int ntx, int nty, int nty_local, int nWorkers, double h, double *u, double *err) {

    extern __shared__ double cache[];

    int tid;
    int i, cacheIndex;
    int l, ll;
    int j, jj0, jj1, jy;

    double x, y;
    double tmp, err_local;


    tid = threadIdx.x + blockIdx.x * blockDim.x;
    cacheIndex = threadIdx.x;
    err_local = 0.0;

    while (tid < nWorkers) {

        jj0 = nty_local * tid;
        jy = jj0 - 2 * tid - 1;

        for(j = 1; j < nty_local-1; j++) {

            y = (double) (jy + j) * h;

            jj1 = (jj0 + j) * ntx;

            for(l = 1; l < ntx-1; l++) {

                x = (double) l * h;

                ll = jj1 + l;

                tmp = fabs(u[ll] - x * y * (x - 1.0) * (y - 1.0));
                if(tmp > err_local) {
                    err_local = tmp;
                }
            }
        }

        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = err_local;
    __syncthreads();

    i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            err_local = cache[cacheIndex];
            tmp = cache[cacheIndex + i];
            if(tmp > err_local) {
                cache[cacheIndex] = tmp;
            }
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        err[blockIdx.x] = cache[0];
    }

}


extern "C" void max_error(int nBlocks, int blockSize, size_t size_err, int ntx, int nty, int nty_local, int nWorkers, double h, double *u, double*err) {

    max_error_kernel<<<nBlocks, blockSize, size_err>>>(ntx, nty, nty_local, nWorkers, h, u, err);

}



