

int main(void) {

    int nx, ny;
    int ntx, nty;
    int n_Workers, nty_local;

    int it, it_max;

    int size_u;
    int n_Blocks, n_Threads;

    double Lx, h;

    double *d_u;
    double *d_f;


    nx = 1024;
    ny = 1024;

    Lx = 1.0;

    h = Lx / double(nx);

    n_Threads = 32;
    n_Blocks = 1;

    n_Workers = n_Threads * n_Blocks;

    nty_local = ny / n_Workers;

    if((nty_local*n_Workers - ny) != 0)
        break;

    ntx = nx + 2;
    nty = ny + 2*n_Workers;

    it_max = 10;

    size_u = ntx * nty * sizeof(double);


    hipMalloc(&d_u, size_u); // solution
    hipMalloc(&d_f, size_u); // source

    // n_Blocks = imin((n_Workers+n_Threads-1) / n_Threads, n_Blocks_max);


    initalize_kernel<<<n_Blocks, n_Threads>>>(ntx, nty_local, n_Workers, h, d_f, d_u);

    it = 0;
    while(it < it_max) {
        it++;
        communication_kernel<<<n_Blocks, n_Threads>>>(ntx, nty_local, n_Workers, d_u);
        compute_kernel<<<n_Blocks, n_Threads>>>(ntx, nty_local, n_Workers, h, d_f, d_u);
    }


    // TODO compare with exact solution
    hipFree(d_u);

    return 0;
}

