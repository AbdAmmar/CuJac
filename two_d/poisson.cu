#include <stdlib.h>
#include <stdio.h>
#include <string.h>


#include "utils.cuh"
#include "init_kernel.cuh"
#include "comp_kernel.cuh"
#include "comm_kernel.cuh"




int main() {


    int n;
    int ntx, nty;
    int nty_local;
    int n_Threads, n_Blocks, n_Workers;

    size_t size_u;
    size_t size_err;

    int i, j, ii, jj, jj0, jj1, l;
    int it, it_max, it_print;

    double L, h;

    double* h_u;
    double* d_u;
    double* d_unew;

    double* d_err;
    double* h_err;
    double err;

    FILE *fptr;
    char readString[100];

    int nDevices;
    hipDeviceProp_t prop;
    hipError_t err_cuda;

    err_cuda = hipGetDeviceCount(&nDevices);
    if(err_cuda != hipSuccess) printf("%s\n", hipGetErrorString(err_cuda));
    for (i = 0; i < nDevices; i++) {
        hipGetDeviceProperties(&prop, i);
        printf("Device: %d/%d\n", i+1, nDevices);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (GHz): %f\n", prop.memoryClockRate/1.0e6);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }

    //n = 4096;
    //n_Threads = 1;
    //n_Blocks = 1;
    fptr = fopen("param.txt", "r");
    if(fptr != NULL) {

        if(fgets(readString, 100, fptr) != NULL) {
            n = atoi(readString);
        } else {
            printf("Not able to read n\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            n_Threads = atoi(readString);
        } else {
            printf("Not able to read n_Threads\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            n_Blocks = atoi(readString);
        } else {
            printf("Not able to read n_Blocks\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            it_max = atoi(readString);
        } else {
            printf("Not able to read it_max\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            it_print = atoi(readString);
        } else {
            printf("Not able to read it_print\n");
        }

    } else {
        printf("Not able to open the file param.txt\n");
    }
    fclose(fptr);


    L = 1.0;
    h = L / (double) (n-1);

    printf("nb on grid points = %d x %d\n", n, n);
    printf("dim of grid = %.1f x %.1f\n", L, L);
    printf("step = %f\n\n", h);


    n_Workers = n_Threads * n_Blocks;
    printf("nb on threads = %d\n", n_Threads);
    printf("nb on blocks = %d\n", n_Blocks);
    printf("nb on workers = %d\n", n_Workers);


    ntx = n;
    nty = n + 2*n_Workers;
    nty_local = n / n_Workers + 2;
    if((nty_local*n_Workers - nty) != 0) {
        printf("Unconsistent dimensions\n");
        exit(0);
    }

    printf("ntx = %d\n", ntx);
    printf("nty = %d\n", nty);
    printf("nty_local = %d\n\n", nty_local);

    size_u = ntx * nty * sizeof(double);
    printf("Size of d_u = %zu Bytes \n\n", size_u);

    size_err = n_Blocks * sizeof(double);
    printf("Size of d_err = %zu Bytes \n\n", size_err);

    checkCudaErrors(hipMalloc(&d_u, size_u), "hipMalloc");
    checkCudaErrors(hipMalloc(&d_unew, size_u), "hipMalloc");
    checkCudaErrors(hipMalloc(&d_err, size_err), "hipMalloc");

    h_u = (double*) malloc(size_u);
    if(h_u == NULL) {
        fprintf(stderr, "Memory allocation failed for h_u\n");
        exit(0);
    }

    h_err = (double*) malloc(size_err);
    if(h_err == NULL) {
        fprintf(stderr, "Memory allocation failed for h_err\n");
        exit(0);
    }


    init<<<n_Blocks, n_Threads>>>(ntx, nty_local, n_Workers, d_u);
    checkCudaErrors(hipGetLastError(), "Kernel init launch failed");
    hipDeviceSynchronize();

    it = 0;
    while(it < it_max) {

        //printf("it = %d/%d\n", it, it_max);

        compute<<<n_Blocks, n_Threads>>>(ntx, nty, nty_local, n_Workers, h, d_u, d_unew);
        checkCudaErrors(hipGetLastError(), "Kernel compute launch failed");
        hipDeviceSynchronize();

        //checkCudaErrors(hipMemcpy(h_u, d_unew, size_u, hipMemcpyDeviceToHost), "hipMemcpy");
        //for(l = 0; l < n_Workers; l++){
        //    jj0 = l * nty_local;
        //    for (j = 1; j < nty_local-1; j++) {
        //        jj1 = (jj0 + j) * ntx;
        //        for (i = 0; i < ntx; i++) {
        //            ii = jj1 + i;
        //            printf("%f  ", h_u[ii]);
        //        }
        //        printf("\n");
        //    }
        //    printf("\n");
        //}
        //printf("\n");

        naivecopy<<<n_Blocks, n_Threads>>>(ntx, nty, nty_local, n_Workers, d_unew, d_u);
        hipDeviceSynchronize();

        communication<<<n_Blocks, n_Threads>>>(ntx, nty_local, n_Workers, d_u);
        hipDeviceSynchronize();

        max_error<<<n_Blocks, n_Threads, size_err>>>(ntx, nty, nty_local, n_Workers, h, d_u, d_err);
        hipDeviceSynchronize();

        hipMemcpy(h_err, d_err, size_err, hipMemcpyDeviceToHost);
        err = h_err[0];
        for (i = 1; i < n_Blocks; i++) {
            if(err < h_err[i]) {
                err = h_err[i];
            }
        }

        if(it%it_print == 0) {
            printf("it = %d/%d, error = %f\n", it, it_max, err);
        }

        it++;
    }


    checkCudaErrors(hipMemcpy(h_u, d_unew, size_u, hipMemcpyDeviceToHost), "hipMemcpy");

    
    hipFree(d_u);
    hipFree(d_unew);
    hipFree(d_err);
    
    for(l = 0; l < n_Workers; l++){
        jj0 = l * nty_local;
        for (j = 1; j < nty_local-1; j++) {
            jj1 = (jj0 + j) * ntx;
            for (i = 0; i < ntx; i++) {
                ii = jj1 + i;
                printf("%f  ", h_u[ii]);
            }
            printf("\n");
        }
        printf("\n");
    }
    printf("\n");

    //printf("%f  \n", h_u[0]);
    free(h_u);
    free(h_err);

    return 0;
}

