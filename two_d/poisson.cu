#include <stdlib.h>
#include <stdio.h>
#include <string.h>


#include "utils.cuh"
#include "init_kernel.cuh"
#include "comp_kernel.cuh"
#include "comm_kernel.cuh"




int main() {


    int n;
    int ntx, nty;
    int nty_local;
    int blockSize, nBlocks, nWorkers;

    size_t size_u;
    size_t size_err;

    int i;
    int it, it_max, it_print;

    double L, h;

    double* d_u;
    double* d_unew;

    double* d_err;
    double* h_err;
    double err;

    FILE *fptr;
    char readString[100];

    int nDevices;
    hipDeviceProp_t prop;



    checkCudaErrors(hipGetDeviceCount(&nDevices), "hipGetDeviceCount)");
    for (i = 0; i < nDevices; i++) {
        hipGetDeviceProperties(&prop, i);
        printf("Device: %d/%d\n", i+1, nDevices);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (GHz): %f\n", prop.memoryClockRate/1.0e6);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Global Memory (GB): %f\n", prop.totalGlobalMem/8.0e9);
        printf("  Constant Memory (Bytes): %zu\n", prop.totalConstMem);
        printf("  Max mem pitch: %ld\n", prop.memPitch);
        printf("  Texture Alignment: %ld\n", prop.textureAlignment);
        printf("  Warp Size : %d\n", prop.warpSize);
        printf("  Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
        printf("  Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("  Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
        printf("  Registers per mp: %d\n", prop.regsPerBlock);
        printf("  Compute capability: %d.%d\n\n", prop.major, prop.minor);
    }


    fptr = fopen("param.txt", "r");
    if(fptr != NULL) {

        if(fgets(readString, 100, fptr) != NULL) {
            n = atoi(readString);
        } else {
            printf("Not able to read n\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            nBlocks = atoi(readString);
        } else {
            printf("Not able to read nBlocks\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            blockSize = atoi(readString);
        } else {
            printf("Not able to read blockSize\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            it_max = atoi(readString);
        } else {
            printf("Not able to read it_max\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            it_print = atoi(readString);
        } else {
            printf("Not able to read it_print\n");
        }

    } else {
        printf("Not able to open the file param.txt\n");
    }
    fclose(fptr);


    L = 1.0;
    h = L / (double) (n-1);

    printf("nb on grid points = %d x %d\n", n, n);
    printf("dim of grid = %.1f x %.1f\n", L, L);
    printf("step = %f\n\n", h);


    nWorkers = nBlocks * blockSize;
    printf("nb of blocks = %d\n", nBlocks);
    printf("size of blocks = %d\n", blockSize);
    printf("nb of workers = %d\n\n", nWorkers);
    if(nWorkers > n) {
        printf("increase n, or decrease nBlocks and/or blockSize");
        exit(0);
    }


    ntx = n;
    nty = n + 2 * nWorkers;
    nty_local = n / nWorkers + 2;

    printf("ntx = %d\n", ntx);
    printf("nty = %d\n", nty);
    printf("nty_local = %d\n\n", nty_local);
    if(nty != nty_local*nWorkers) {
        printf("bad set of parameters !");
        exit(0);
    }



    size_u = ntx * nty * sizeof(double);
    printf("Size of d_u = %zu Bytes \n\n", size_u);

    size_err = nBlocks * sizeof(double);
    printf("Size of d_err = %zu Bytes \n\n", size_err);

    checkCudaErrors(hipMalloc(&d_u, size_u), "hipMalloc");
    checkCudaErrors(hipMalloc(&d_unew, size_u), "hipMalloc");
    checkCudaErrors(hipMalloc(&d_err, size_err), "hipMalloc");

    h_err = (double*) malloc(size_err);
    if(h_err == NULL) {
        fprintf(stderr, "Memory allocation failed for h_err\n");
        exit(0);
    }


    init<<<nBlocks, blockSize>>>(ntx, nty_local, nWorkers, d_u);
    hipDeviceSynchronize();

    it = 1;
    while(it <= it_max) {

        if(it%2 != 0) {
            compute<<<nBlocks, blockSize>>>(ntx, nty, nty_local, nWorkers, h, d_u, d_unew);
            hipDeviceSynchronize();
            communication<<<nBlocks, blockSize>>>(ntx, nty_local, nWorkers, d_unew);
            hipDeviceSynchronize();
        } else {
            compute<<<nBlocks, blockSize>>>(ntx, nty, nty_local, nWorkers, h, d_unew, d_u);
            hipDeviceSynchronize();
            communication<<<nBlocks, blockSize>>>(ntx, nty_local, nWorkers, d_u);
            hipDeviceSynchronize();
        }

        if(it%it_print == 0) {
            max_error<<<nBlocks, blockSize, size_err>>>(ntx, nty, nty_local, nWorkers, h, d_u, d_err);
            hipDeviceSynchronize();
            checkCudaErrors(hipMemcpy(h_err, d_err, size_err, hipMemcpyDeviceToHost), "hipMemcpy");
            err = h_err[0];
            for (i = 1; i < nBlocks; i++) {
                if(err < h_err[i]) {
                    err = h_err[i];
                }
            }
            printf("it = %d/%d, error = %f\n", it, it_max, err);
        }

        it++;
    }

    //int l, j, ii, jj0, jj1;
    //double* h_u;
    //h_u = (double*) malloc(size_u);
    //if(h_u == NULL) {
    //    fprintf(stderr, "Memory allocation failed for h_u\n");
    //    exit(0);
    //}
    //checkCudaErrors(hipMemcpy(h_u, d_unew, size_u, hipMemcpyDeviceToHost), "hipMemcpy");
    //for(l = 0; l < nWorkers; l++){
    //    jj0 = l * nty_local;
    //    for (j = 1; j < nty_local-1; j++) {
    //        jj1 = (jj0 + j) * ntx;
    //        for (i = 0; i < ntx; i++) {
    //            ii = jj1 + i;
    //            printf("%f  ", h_u[ii]);
    //        }
    //        printf("\n");
    //    }
    //    printf("\n");
    //}
    //printf("\n");
    //free(h_u);


    free(h_err);


    hipFree(d_u);
    hipFree(d_unew);
    hipFree(d_err);

    return 0;
}

