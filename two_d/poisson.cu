#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>


#include "utils.cuh"
#include "init_kernel.cuh"

//__global__ void init(int ntx, int nty_local, int n_Workers, double *u) {
//
//
//    int tid;
//    int j, jj0, jj1, l;
//
//    tid = threadIdx.x + blockIdx.x * blockDim.x;
//
//    //while (tid < n_Workers) {
//    while (tid < 1) {
//
//        jj0 = nty_local * tid;
//
//        for(j = 0; j < nty_local; j++) {
//
//            jj1 = (jj0 + j) * ntx;
//
//            for(l = 0; l < ntx; l++) {
//
//                u[l + jj1] = 1.0;
//
//            }
//        }
//
//        tid += blockDim.x * gridDim.x;
//    }
//}




int main() {

    int n;
    int ntx, nty;
    int nty_local;
    int n_Threads, n_Blocks, n_Workers;

    size_t size_u;

    int i, j, ii, jj, jj0, jj1, l;
    int it, it_max, it_print;

    double L, h;

    double* h_u;
    double* d_u;

    double* d_err_i;
    double err;

    FILE *fptr;
    char readString[100];

    n = 4096;
    n_Threads = 1;
    n_Blocks = 1;
    //fptr = fopen("param.txt", "r");
    //if(fptr != NULL) {

    //    if(fgets(readString, 100, fptr) != NULL) {
    //        n = atoi(readString);
    //    } else {
    //        printf("Not able to read n\n");
    //    }

    //    if(fgets(readString, 100, fptr) != NULL) {
    //        n_Threads = atoi(readString);
    //    } else {
    //        printf("Not able to read n_Threads\n");
    //    }

    //    if(fgets(readString, 100, fptr) != NULL) {
    //        n_Blocks = atoi(readString);
    //    } else {
    //        printf("Not able to read n_Blocks\n");
    //    }

    //    if(fgets(readString, 100, fptr) != NULL) {
    //        it_max = atoi(readString);
    //    } else {
    //        printf("Not able to read it_max\n");
    //    }

    //    if(fgets(readString, 100, fptr) != NULL) {
    //        it_print = atoi(readString);
    //    } else {
    //        printf("Not able to read it_print\n");
    //    }

    //} else {
    //    printf("Not able to open the file param.txt\n");
    //}
    //fclose(fptr);


    L = 1.0;
    h = L / (double) (n-1);

    //printf("nb on grid points = %d x %d\n", n, n);
    //printf("dim of grid = %.1f x %.1f\n", L, L);
    //printf("step = %f\n\n", h);


    n_Workers = n_Threads * n_Blocks;
    //printf("nb on threads = %d\n", n_Threads);
    //printf("nb on blocks = %d\n", n_Blocks);
    //printf("nb on workers = %d\n", n_Workers);


    ntx = n;
    nty = n + 2*n_Workers;
    nty_local = n / n_Workers + 2;
    //if((nty_local*n_Workers - nty) != 0) {
    //    printf("Unconsistent dimensions\n");
    //    exit(0);
    //}

    //printf("ntx = %d\n", ntx);
    //printf("nty = %d\n", nty);
    //printf("nty_local = %d\n\n", nty_local);

    size_u = ntx * nty * sizeof(double);
    //printf("Size of d_u = %zu Bytes \n\n", size_u);


    checkCudaErrors(hipMalloc(&d_u, size_u), "hipMalloc");
    //hipMalloc(&d_err_i, n_Workers * sizeof(double));

    h_u = (double*) malloc(size_u);
    if(h_u == NULL) {
        fprintf(stderr, "Memory allocation failed for h_u\n");
        exit(0);
    }


    init<<<n_Blocks, n_Threads>>>(ntx, nty_local, n_Workers, d_u);
    checkCudaErrors(hipGetLastError(), "Kernel launch failed");

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError(), "Kernel launch failed");

    //it = 0;
    //while(it < it_max) {
    //    compute_kernel<<<n_Blocks, n_Threads>>>(ntx, nty, nty_local, h, d_u);
    //    it++;
    //}


  
    checkCudaErrors(hipMemcpy(h_u, d_u, size_u, hipMemcpyDeviceToHost), "hipMemcpy");

    
    hipFree(d_u);
    //hipFree(d_err_i);
    
    //for(l = 0; l < n_Workers; l++){
    //    jj0 = l * nty_local;
    //    for (j = 1; j < nty_local-1; j++) {
    //        jj1 = (jj0 + j) * ntx;
    //        for (i = 0; i < ntx; i++) {
    //            ii = jj1 + i;
    //            printf("%f  ", h_u[ii]);
    //        }
    //        printf("\n");
    //    }
    //    printf("\n");
    //}
    //printf("\n");

    printf("%f  \n", h_u[0]);
    free(h_u);


    return 0;
}

