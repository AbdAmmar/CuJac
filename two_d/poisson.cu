#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>


#include "utils.cuh"
#include "init_kernel.cuh"
#include "comp_kernel.cuh"
#include "comm_kernel.cuh"




int main() {


    int n;
    int ntx, nty;
    int nty_local;
    int blockSize, n_Blocks, streamSize, nWorkers, nStream;

    size_t size_u;
    size_t size_err;

    int i;
    int it, it_max, it_print;

    double L, h;

    double* d_u;
    double* d_unew;

    double* d_err;
    double* h_err;
    double err;

    FILE *fptr;
    char readString[100];

    int nDevices;
    hipDeviceProp_t prop;
    hipError_t err_cuda;



    err_cuda = hipGetDeviceCount(&nDevices);
    if(err_cuda != hipSuccess) printf("%s\n", hipGetErrorString(err_cuda));
    for (i = 0; i < nDevices; i++) {
        hipGetDeviceProperties(&prop, i);
        printf("Device: %d/%d\n", i+1, nDevices);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (GHz): %f\n", prop.memoryClockRate/1.0e6);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }


    fptr = fopen("param.txt", "r");
    if(fptr != NULL) {

        if(fgets(readString, 100, fptr) != NULL) {
            n = atoi(readString);
        } else {
            printf("Not able to read n\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            nStream = atoi(readString);
        } else {
            printf("Not able to read nStream\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            streamSize = atoi(readString);
        } else {
            printf("Not able to read streamSize\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            blockSize = atoi(readString);
        } else {
            printf("Not able to read blockSize\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            it_max = atoi(readString);
        } else {
            printf("Not able to read it_max\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            it_print = atoi(readString);
        } else {
            printf("Not able to read it_print\n");
        }

    } else {
        printf("Not able to open the file param.txt\n");
    }
    fclose(fptr);


    L = 1.0;
    h = L / (double) (n-1);

    printf("nb on grid points = %d x %d\n", n, n);
    printf("dim of grid = %.1f x %.1f\n", L, L);
    printf("step = %f\n\n", h);


    n_Blocks = (streamSize + blockSize - 1) / blockSize;
    nWorkers = min(streamSize * nStream, (n+2)/3);
    printf("nb of streams = %d\n", nStream);
    printf("size of stream = %d\n", streamSize);
    if(nStream*streamSize > n) {
        printf("increase n, or decrease nStream and/or streamSize");
        exit(0);
    }
    printf("size of blocks = %d\n", blockSize);
    printf("nb of blocks = %d\n", n_Blocks);
    printf("nb of workers = %d\n\n", nWorkers);


    ntx = n;
    nty = n + 2 * nWorkers;
    nty_local = n / nWorkers + 2;

    printf("ntx = %d\n", ntx);
    printf("nty = %d\n", nty);
    printf("nty_local = %d\n\n", nty_local);
    if(nty != nty_local*nWorkers) {
        printf("bad set of parameters !");
        exit(0);
    }



    size_u = ntx * nty * sizeof(double);
    printf("Size of d_u = %zu Bytes \n\n", size_u);

    size_err = n_Blocks * sizeof(double);
    printf("Size of d_err = %zu Bytes \n\n", size_err);

    checkCudaErrors(hipMalloc(&d_u, size_u), "hipMalloc");
    checkCudaErrors(hipMalloc(&d_unew, size_u), "hipMalloc");
    checkCudaErrors(hipMalloc(&d_err, size_err), "hipMalloc");

    h_err = (double*) malloc(size_err);
    if(h_err == NULL) {
        fprintf(stderr, "Memory allocation failed for h_err\n");
        exit(0);
    }


    init<<<n_Blocks, blockSize>>>(ntx, nty_local, nWorkers, d_u);
    hipDeviceSynchronize();

    it = 1;
    while(it <= it_max) {

        //compute<<<n_Blocks, blockSize>>>(ntx, nty, nty_local, nWorkers, h, d_u, d_unew);
        //hipDeviceSynchronize();
        //naivecopy<<<n_Blocks, blockSize>>>(ntx, nty, nty_local, nWorkers, d_unew, d_u);
        //hipDeviceSynchronize();
        //communication<<<n_Blocks, blockSize>>>(ntx, nty_local, nWorkers, d_u);
        //hipDeviceSynchronize();

        if(it%2 != 0) {
            compute<<<n_Blocks, blockSize>>>(ntx, nty, nty_local, nWorkers, h, d_u, d_unew);
            hipDeviceSynchronize();
            communication<<<n_Blocks, blockSize>>>(ntx, nty_local, nWorkers, d_unew);
            hipDeviceSynchronize();
        } else {
            compute<<<n_Blocks, blockSize>>>(ntx, nty, nty_local, nWorkers, h, d_unew, d_u);
            hipDeviceSynchronize();
            communication<<<n_Blocks, blockSize>>>(ntx, nty_local, nWorkers, d_u);
            hipDeviceSynchronize();
        }

        if(it%it_print == 0) {
            max_error<<<n_Blocks, blockSize, size_err>>>(ntx, nty, nty_local, nWorkers, h, d_u, d_err);
            hipDeviceSynchronize();
            hipMemcpy(h_err, d_err, size_err, hipMemcpyDeviceToHost);
            err = h_err[0];
            for (i = 1; i < n_Blocks; i++) {
                if(err < h_err[i]) {
                    err = h_err[i];
                }
            }
            printf("it = %d/%d, error = %f\n", it, it_max, err);
        }

        it++;
    }

    //int l, j, ii, jj0, jj1;
    //double* h_u;
    //h_u = (double*) malloc(size_u);
    //if(h_u == NULL) {
    //    fprintf(stderr, "Memory allocation failed for h_u\n");
    //    exit(0);
    //}
    //checkCudaErrors(hipMemcpy(h_u, d_unew, size_u, hipMemcpyDeviceToHost), "hipMemcpy");
    //for(l = 0; l < nWorkers; l++){
    //    jj0 = l * nty_local;
    //    for (j = 1; j < nty_local-1; j++) {
    //        jj1 = (jj0 + j) * ntx;
    //        for (i = 0; i < ntx; i++) {
    //            ii = jj1 + i;
    //            printf("%f  ", h_u[ii]);
    //        }
    //        printf("\n");
    //    }
    //    printf("\n");
    //}
    //printf("\n");
    //free(h_u);


    free(h_err);

    hipFree(d_u);
    hipFree(d_unew);
    hipFree(d_err);

    return 0;
}

