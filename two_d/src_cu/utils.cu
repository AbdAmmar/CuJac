

#include <hip/hip_runtime.h>
#include <stdio.h>


void check_Errors() {
    hipError_t err = hipGetLastError();
    printf("CUDA Error ? %s\n", hipGetErrorString(err));
}

void checkCudaErrors(hipError_t err, const char* msg, const char* file, int line) {
    if (err != hipSuccess) {
        printf("CUDA Error in %s at line %d\n", file, line);
        printf("%s - %s\n", msg, hipGetErrorString(err));
        exit(0);
    }
}


extern "C" void checkCudaErrors_C(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        printf("CUDA Error: %s - %s", msg, hipGetErrorString(err));
        exit(0);
    }
}


__global__ void max_error_kernel(int ntx, int nty, int nty_local, int nWorkers, double h, double *u, double *err) {

    extern __shared__ double cache[];

    int tid;
    int i, cacheIndex;
    int l, ll;
    int j, jj0, jj1;

    double x, y;
    double tmp, err_local;


    tid = threadIdx.x + blockIdx.x * blockDim.x;
    cacheIndex = threadIdx.x;
    err_local = 0.0;

    while (tid < nWorkers) {

        jj0 = nty_local * tid;

        for(j = 0; j < nty_local; j++) {

            y = (double) (jj0 + j) * h;

            jj1 = (jj0 + j) * ntx;

            for(l = 0; l < ntx; l++) {

                x = (double) l * h;

                ll = jj1 + l;

                tmp = fabs(u[ll] - x * y * (x - 1.0) * (y - 1.0));
                if(tmp > err_local) {
                    err_local = tmp;
                }
            }
        }

        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = err_local;
    __syncthreads();

    i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            err_local = cache[cacheIndex];
            tmp = cache[cacheIndex + i];
            if(tmp > err_local) {
                cache[cacheIndex] = tmp;
            }
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        err[blockIdx.x] = cache[0];
    }

}


extern "C" void max_error(int nBlocks, int blockSize, size_t size_err, int ntx, int nty, int nty_local, int nWorkers, double h, double *u, double*err) {

    max_error_kernel<<<nBlocks, blockSize, size_err>>>(ntx, nty, nty_local, nWorkers, h, u, err);

}


