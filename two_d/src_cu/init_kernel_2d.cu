
#include <hip/hip_runtime.h>

__global__ void init_2d_kernel(int ntx, int ntx_local, int nty_local, int nWorkers_x, int nWorkers_y, double *u) {


    int tid_x;
    int tid_y;

    int j, jj0, jj1;
    int i, ii0;
  
    int js, je;
    int is, ie;

    tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    tid_y = threadIdx.y + blockIdx.y * blockDim.y;

    while (tid_y < nWorkers_y) {

        jj0 = tid_y * nty_local;
        js = jj0;
        je = jj0 + nty_local;

        while (tid_x < nWorkers_x) {

            ii0 = tid_x * ntx_local;
            is = ii0;
            ie = ii0 + ntx_local;

            for(j = js; j < je; j++) {

                jj1 = j * ntx;

                for(i = is; i < ie; i++) {

                    u[i + jj1] = 0.0;

                }
            }

            tid_x += blockDim.x * gridDim.x;

        }

        tid_y += blockDim.y * gridDim.y;
    }
}


extern "C" void init_2d(dim3 grid_dim, dim3 block_dim,
                        int ntx, int ntx_local, int nty_local, int nWorkers_x, int nWorkers_y, double *u) {

    init_2d_kernel<<<grid_dim, block_dim>>>(ntx, ntx_local, nty_local, nWorkers_x, nWorkers_y, u);

}

