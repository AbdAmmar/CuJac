
#include <hip/hip_runtime.h>



__global__ void compute_kernel(int ntx, int nty, int nty_local, int nWorkers, double h, double *u_old, double *u_new) {

    int tid;

    int l, ll;
    int js, je, jj_check;
    int j, jj0, jj1;

    //int do_bc_top;
    //int do_bc_bottom;

    double x, y, y_tmp;

    const double h_ct = 2.0 * h * h;

    tid = threadIdx.x + blockIdx.x * blockDim.x;
  
    //do_bc_top = 0;
    //do_bc_bottom = 0;

    if(tid != 0) {
        js = 0;
    } else {
        js = 1;
        //do_bc_top = 1;
    }

    jj_check = nty - nty_local;
    //jf = ntx * nty;

    while (tid < nWorkers) {

        jj0 = nty_local * tid;

        if(jj0 != jj_check) {
            je = nty_local;
        } else {
            je = nty_local - 1;
            //do_bc_bottom = 1;
        }

        for(j = js; j < je; j++) {
        
            y = (double) (jj0 + j) * h;
            y_tmp = y * (y - 1.0);
        
            jj1 = (jj0 + j) * ntx;
        
            for(l = 1; l < ntx-1; l++) {
        
                x = (double) l * h;
        
                ll = jj1 + l;
        
                u_new[ll] = 0.25 * ( u_old[ll - 1] + u_old[ll + 1] + u_old[ll - ntx] + u_old[ll + ntx] 
                                   - h_ct * (x * (x - 1.0) + y_tmp) ) ;
            }
        }
        
        tid += blockDim.x * gridDim.x;
    }

    // TODO
    //if(do_bc_top == 1) {
    //    for(l = 0; l < ntx; l++) {
    //        u_new[l] = 0.0;
    //    }
    //}
    //if(do_bc_bottom = 1) {
    //    for(l = 0; l < ntx; l++) {
    //        u_new[jf - l - 1] = 0.0;
    //    }
    //}

}

extern "C" void compute(int nBlocks, int blockSize, int ntx, int nty, int nty_local, int nWorkers, double h, double *u_old, double *u_new) {

    compute_kernel<<<nBlocks, blockSize>>>(ntx, nty, nty_local, nWorkers, h, u_old, u_new);

}

