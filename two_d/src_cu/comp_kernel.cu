
#include <hip/hip_runtime.h>



__global__ void compute_kernel(int ntx, int nty, int nty_local, int nWorkers, double h, double *u_old, double *u_new) {

    int tid;

    int l, ll;
    int js, je, jj_check, jf;
    int j, jj0, jj1;

    double x, y, y_tmp;

    const double h_ct = 2.0 * h * h;

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid != 0) {
        js = 0;
    } else {
        js = 1;
        for(l = 0; l < ntx; l++) {
            u_new[l] = 0.0;
        }
    }

    jj_check = nty - nty_local;
    jf = ntx * nty;

    while (tid < nWorkers) {

        jj0 = nty_local * tid;

        if(jj0 != jj_check) {
            je = nty_local;
        } else {
            je = nty_local - 1;
            for(l = 0; l < ntx; l++) {
                u_new[jf - l - 1] = 0.0;
            }
        }

        for(j = js; j < je; j++) {
        
            y = (double) (jj0 + j) * h;
            y_tmp = y * (y - 1.0);
        
            jj1 = (jj0 + j) * ntx;
        
            for(l = 1; l < ntx-1; l++) {
        
                x = (double) l * h;
        
                ll = jj1 + l;
        
                u_new[ll] = 0.25 * ( u_old[ll - 1] + u_old[ll + 1] + u_old[ll - ntx] + u_old[ll + ntx] 
                                   - h_ct * (x * (x - 1.0) + y_tmp) ) ;
            }
        }
        
        tid += blockDim.x * gridDim.x;
    }
}

extern "C" void compute(int nBlocks, int blockSize, int ntx, int nty, int nty_local, int nWorkers, double h, double *u_old, double *u_new) {

    compute_kernel<<<nBlocks, blockSize>>>(ntx, nty, nty_local, nWorkers, h, u_old, u_new);

}

