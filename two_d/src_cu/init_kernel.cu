
#include <hip/hip_runtime.h>

__global__ void init_kernel(int ntx, int nty_local, int nWorkers, double *u) {


    int tid;
    int j, jj1, l;

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < nWorkers) {

        for(j = 0; j < nty_local; j++) {

            jj1 = j * ntx;

            for(l = 0; l < ntx; l++) {

                u[l + jj1] = 0.0;

            }
        }

        tid += blockDim.x * gridDim.x;
    }
}


extern "C" void init(int nBlocks, int blockSize, int ntx, int nty_local, int nWorkers, double *u) {

    init_kernel<<<nBlocks, blockSize>>>(ntx, nty_local, nWorkers, u);

}

