
#include <hip/hip_runtime.h>



__global__ void compute_2d_kernel(int ntx, int nty, int ntx_local, int nty_local, int nWorkers_x, int nWorkers_y, double h, double *u_old, double *u_new) {

    int tid_x, tid_y;

    int js, je, jj_check;
    int is, ie, ii_check;

    int j, jj0, jj1;
    int i, ii0;
    int ll;

    //int do_bc_top;
    //int do_bc_bottom;
    //int do_bc_left;
    //int do_bc_right;

    double x, y, y_tmp;

    const double h_ct = 2.0 * h * h;

    tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    tid_y = threadIdx.y + blockIdx.y * blockDim.y;

    //do_bc_top = 0;
    //do_bc_bottom = 0;
    //do_bc_left = 0;
    //do_bc_right = 0;

    if(tid_y != 0) {
        js = 0;
    } else {
        js = 1;
        //do_bc_top = 1;
    }

    if(tid_x != 0) {
        is = 0;
    } else {
        is = 1;
        //do_bc_left = 1;
    }

    jj_check = nty - nty_local;
    //j_f = ntx * nty;

    ii_check = ntx - ntx_local;
    //i_f = ntx * nty;

    while (tid_y < nWorkers_y) {

        jj0 = nty_local * tid_y;

        if(jj0 != jj_check) {
            je = nty_local;
        } else {
            je = nty_local - 1;
            //do_bc_bottom = 1;
        }

        while (tid_x < nWorkers_x) {

            ii0 = ntx_local * tid_x;

            if(ii0 != ii_check) {
                ie = ntx_local;
            } else {
                ie = ntx_local - 1;
                //do_bc_right = 1;
            }

    
            for(j = jj0 + js; j < jj0 + je; j++) {
            
                y = (double) j * h;
                y_tmp = y * (y - 1.0);
            
                jj1 = j * ntx;
            
                for(i = ii0 + is; i < ii0 + ie; i++) {
            
                    x = (double) i * h;
            
                    ll = jj1 + i;
            
                    u_new[ll] = 0.25 * ( u_old[ll - 1] + u_old[ll + 1] + u_old[ll - ntx] + u_old[ll + ntx] 
                                       - h_ct * (x * (x - 1.0) + y_tmp) ) ;
                }
            }
            
            tid_x += blockDim.x * gridDim.x;
        }

        tid_y += blockDim.y * gridDim.y;
    }

    // TODO
    //if(do_bc_top == 1) {
    //    for(l = 0; l < ntx_local; l++) {
    //        u_new[l] = 0.0;
    //    }
    //}

    //if(do_bc_bottom = 1) {
    //    for(l = 0; l < ntx; l++) {
    //        u_new[j_f - l - 1] = 0.0;
    //    }
    //}

}



extern "C" void compute_2d(dim3 grid_dim, dim3 block_dim, 
                           int ntx, int nty, int ntx_local, int nty_local, int nWorkers_x, int nWorkers_y, double h, double *u_old, double *u_new) {

    compute_2d_kernel<<<grid_dim, block_dim>>>(ntx, nty, ntx_local, nty_local, nWorkers_x, nWorkers_y, h, u_old, u_new);

}

