
#include <hip/hip_runtime.h>

__global__ void communication_kernel(int ntx, int nty_local, int nWorkers, double *u) {


    int tid;
    int l;
    int jj0, jj1, jj2;
    double tmp;

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < nWorkers) {

        jj0 = nty_local * tid;

        if(jj0 != 0) {
            jj1 = (jj0 - 1) * ntx;
            jj2 = jj1 + ntx;
            for(l = 0; l < ntx; l++) {
                tmp = u[jj1 + l];
                u[jj1 + l] = u[jj2 + l];
                u[jj2 + l] = tmp;
            }
        }

        tid += blockDim.x * gridDim.x;
    }
}



extern "C" void communication(int nBlocks, int blockSize, int ntx, int nty_local, int nWorkers, double *u) {

    communication_kernel<<<nBlocks, blockSize>>>(ntx, nty_local, nWorkers, u);


}




