
#include <hip/hip_runtime.h>

__global__ void initalize_kernel(int ntx, int nty_local, int n_Workers, double h, double *f, double *u) {


    int tid;
    int j, k, l;
    int ii0, ii1, ii2;
    int ll0, ll1, ll2, ll3;
    double x, y;

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < n_Workers) {

        ii0 = nty_local * tid;
        ii1 = ii0 + nty_local + 1;
        ii2 = 2 * tid;

        for(j = 1; j <= nty_local; j++) {

            k = ii0 + j;
            y = __int2double_rn(k) * h;

            ll0 = (k + ii2) * ntx;
            for(l = 0; l < ntx; l++) {

                ll1 = l + ll0;

                x = __int2double_rn(l+1) * h;

                // TODO : call a general function for the source
                f[ll1] = 2.0 * (x * x - x + y * y - y);
                u[ll1] = 0.0;
            }

            ll0 = ii0 + ii2;
            ll1 = ii1 + ii2;
            for(l = 0; l < ntx; l++) {

                ll2 = l + ll0;
                ll3 = l + ll1;

                f[ll2] = f[ll2+1];
                f[ll3] = f[ll3-1];
               
                u[ll2] = u[ll2+1];
                u[ll3] = u[ll3-1];
            }
        }

        tid += blockDim.x * gridDim.x;
    }
}

