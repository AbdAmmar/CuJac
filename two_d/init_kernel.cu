
#include <hip/hip_runtime.h>

__global__ void init(int ntx, int nty_local, int n_Workers, double *u) {


    int tid;
    int j, jj1, l;

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < n_Workers) {

        for(j = 0; j < nty_local; j++) {

            jj1 = j * ntx;

            for(l = 0; l < ntx; l++) {

                u[l + jj1] = 0.0;

            }
        }

        tid += blockDim.x * gridDim.x;
    }
}

