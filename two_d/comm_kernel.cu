#include "hip/hip_runtime.h"

__global__ void communication_kernel(int ntx, int nty_local, int n_Workers, double *u) {


    int tid;
    int l;
    int ii, ll;
    double tmp;

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < n_Workers) {

        ii = nty_local * tid;

        for(l = 0; l < ntx; l++) {

            ll = ii + l

            tmp = u[ll];
            u[ll] = u[ll-1];
            u[ll-1] = tmp;
        }

        tid += blockDim.x * gridDim.x;
    }
}

