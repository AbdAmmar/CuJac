
#include <hip/hip_runtime.h>

__global__ void communication(int ntx, int nty_local, int nWorkers, double *u) {


    int tid;
    int l;
    int jj0, jj1, jj2;
    double tmp;

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < nWorkers) {

        jj0 = nty_local * tid;

        if(jj0 != 0) {
            jj1 = (jj0 - 1) * ntx;
            jj2 = jj1 + ntx;
            for(l = 0; l < ntx; l++) {
                tmp = u[jj1 + l];
                u[jj1 + l] = u[jj2 + l];
                u[jj2 + l] = tmp;
            }
        }

        tid += blockDim.x * gridDim.x;
    }
}



__global__ void naivecopy(int ntx, int nty, int nty_local, int nWorkers, double*u_old, double *u_new) {

    int tid;
    int l, ll;
    int j, jj0, jj1;

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < nWorkers) {

        jj0 = nty_local * tid;

        for(j = 0; j < nty_local; j++) {
            jj1 = (jj0 + j) * ntx;
            for(l = 0; l < ntx; l++) {
                ll = jj1 + l;
                u_new[ll] = u_old[ll];
            }
        }

        tid += blockDim.x * gridDim.x;
    }
}





