
#include <hip/hip_runtime.h>

__global__ void communication(int ntx, int nty_local, int n_Workers, double *u) {


    int tid;
    int l;
    int jj0, jj1, jj2;
    double tmp;

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < n_Workers) {

        jj0 = nty_local * tid;

        if(jj0 != 0) {
            jj1 = (jj0 - 1) * ntx;
            jj2 = jj1 + ntx;
            for(l = 0; l < ntx; l++) {
                tmp = u[jj1 + l];
                u[jj1 + l] = u[jj2 + l];
                u[jj2 + l] = tmp;
            }
        }

        tid += blockDim.x * gridDim.x;
    }
}



__global__ void naivecopy(int ntx, int nty, int nty_local, int n_Workers, double*u_new, double *u) {

    int tid;
    int l, ll;
    int j, jj0, jj1;

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < n_Workers) {

        jj0 = nty_local * tid;

        for(j = 0; j < nty_local; j++) {
            jj1 = (jj0 + j) * ntx;
            for(l = 0; l < ntx; l++) {
                ll = jj1 + l;
                u[ll] = u_new[ll];
            }
        }

        tid += blockDim.x * gridDim.x;
    }
}





