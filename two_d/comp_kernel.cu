
#include <hip/hip_runtime.h>



__global__ void compute(int ntx, int nty, int nty_local, int n_Workers, double h, double *u, double *u_new) {

    int tid;

    int l, ll;
    int j, jj0, jj1, jj2, jy;

    double x, y, h2;

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    h2 = h * h;

    while (tid < n_Workers) {

        jj0 = nty_local * tid;

        jy = jj0 - 2 * tid - 1;

        for(j = 1; j < nty_local-1; j++) {
        
            y = (double) (jy + j) * h;
        
            jj1 = (jj0 + j) * ntx;
        
            for(l = 1; l < ntx-1; l++) {
        
                x = (double) l * h;
        
                ll = jj1 + l;
        
                u_new[ll] = 0.25 * (u[ll - 1] + u[ll + 1] + u[ll - ntx] + u[ll + ntx] - h2 * (2.0 * (x * (x - 1.0) + y * (y - 1.0))));
            }
        }
        
        if(jj0 != 0) {
            jj1 = (jj0 + 1) * ntx;
            jj2 = jj1 - ntx;
            for(l = 0; l < ntx; l++) {
                u_new[jj2 + l] = u_new[jj1 + l];
            }
        } else {
            for(l = 0; l < ntx; l++) {
                u_new[ntx + l] = 0.0;
            }
        }
      
        if(jj0 + nty_local != nty) {
            jj1 = (jj0 + nty_local) * ntx;
            jj2 = jj1 - ntx;
            for(l = 0; l < ntx; l++) {
                u_new[jj1 - l - 1] = u_new[jj2 - l - 1];
            }
        } else {
            jj1 = (nty - 1) * ntx;
            for(l = 0; l < ntx; l++) {
                u_new[jj1 - l - 1] = 0.0;
            }
        }

        tid += blockDim.x * gridDim.x;
    }
}

