#include "hip/hip_runtime.h"

__global__ void compute_kernel(int ntx, int nty_local, int n_Workers, double h, double *f, double *u) {

    int tid;
    int l;
    int ii, ll;
    double tmp;

    double *u_tmp;
    size_t size_tmp;

    size_tmp = ntx * nty_local * sizeof(double);

    hipMalloc(&u_tmp, size_tmp);

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < n_Workers) {

        ii = nty_local* tid;

        for(j = 0; j < nty_local; j++) {

            jj0 = ntx * j;
            jj1 = jj0 + ii;

            for(l = 0; l < ntx; l++) {

                ll0 = l + jj0;
                ll1 = l + jj1;

                u_tmp[ll0] = u[ll1];
            }
        }

        for(j = 1; j < nty_local-1; j++) {

            jj0 = ntx * j;
            jj1 = jj0 + ii;

            for(l = 1; l < ntx-1; l++) {

                ll0 = l + jj0;
                ll1 = l + jj1;

                u[ll1] = 0.25 * ( u_tmp[(l - 1) + ntx * j] + u_tmp[(l + 1) + ntx * j]
                                  u_tmp[l + ntx * (j - 1)] + u_tmp[l + ntx * (j + 1)] ) 
                       - h * h * f[ll1];
            }
        }

        tid += blockDim.x * gridDim.x;
    }
}

