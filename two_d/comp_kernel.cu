
#include <hip/hip_runtime.h>



__global__ void compute_kernel(int ntx, int nty, int nty_local, int n_Workers, double h, double *u) {

    int tid;

    int l, ll;
    int j, jj0, jj1, jj2, jy;

    double *u_tmp;
    size_t size_tmp;

    double x, y, h2;

    tid = threadIdx.x + blockIdx.x * blockDim.x;

    h2 = h * h;

    size_tmp = ntx * nty_local * sizeof(double);

    while (tid < n_Workers) {

        // TODO
        // single big allocate ?
        hipMalloc(&u_tmp, size_tmp);

        jj0 = nty_local * tid;

        for(j = 0; j < nty_local; j++) {

            jj1 = j * ntx;
            jj2 = (jj0 + j) * ntx;

            for(l = 0; l < ntx; l++) {

                u_tmp[jj1 + l] = u[jj2 + l];

            }
        }

        jy = jj0 - 2 * tid - 1;

        for(j = 1; j < nty_local-1; j++) {
        
            y = (double) (jy + j) * h;
        
            jj1 = j * ntx;
            jj2 = (jj0 + j) * ntx;
        
            for(l = 1; l < ntx-1; l++) {
        
                x = (double) l * h;
        
                ll = jj1 + l;
        
                u[jj2 + l] = 0.25 * ( u_tmp[ll - 1] + u_tmp[ll + 1] + u_tmp[ll - ntx] + u_tmp[ll + ntx] 
                                    - h2 * (2.0 * (x * (x - 1) + y * (y - 1))) );
            }
        }
        
        free(u_tmp);

        if(jj0 != 0) {
            jj1 = (jj0 + 1) * ntx;
            jj2 = jj1 - ntx;
            for(l = 0; l < ntx; l++) {
                u[jj2 + l] = u[jj1 + l];
            }
        } else {
            for(l = 0; l < ntx; l++) {
                u[ntx + l] = 0.0;
            }
        }
      
        if(jj0 + nty_local != nty) {
            jj1 = (jj0 + nty_local) * ntx;
            jj2 = jj1 - ntx;
            for(l = 0; l < ntx; l++) {
                u[jj1 - l - 1] = u[jj2 - l - 1];
            }
        } else {
            jj1 = (nty - 1) * ntx;
            for(l = 0; l < ntx; l++) {
                u[jj1 - l - 1] = 0.0;
            }
        }

        tid += blockDim.x * gridDim.x;
    }
}

