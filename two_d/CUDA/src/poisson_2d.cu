#include <stdlib.h>
#include <stdio.h>
#include <string.h>


#include "utils.cuh"
#include "init_kernel_2d.cuh"
#include "comp_kernel_2d.cuh"




int main() {


    int n;
    int ntx, ntx_local;
    int nty, nty_local;
    int blockxSize, nxBlocks, nWorkers_x;
    int blockySize, nyBlocks, nWorkers_y;

    size_t size_u;
    size_t size_err;

    int i;
    int it, it_max, it_print;

    double L, h;

    double* d_u;
    double* d_unew;

    double* d_err;
    double* h_err;
    double err;

    FILE *fptr;
    char readString[100];

    int nDevices;
    hipDeviceProp_t prop;

    hipEvent_t start, stop;
    float tt;

    checkCudaErrors(hipEventCreate(&start), "hipEventCreate", __FILE__, __LINE__);
    checkCudaErrors(hipEventCreate(&stop), "hipEventCreate",  __FILE__, __LINE__);

    checkCudaErrors(hipEventRecord(start), "hipEventRecord", __FILE__, __LINE__);

    nDevices = 0;
    checkCudaErrors(hipGetDeviceCount(&nDevices), "hipGetDeviceCount)", __FILE__, __LINE__);
    if(nDevices == 0) {
        printf("no available GPU(s)\n");
        exit(0);
    } else {
        printf("Detected %d GPU(s)\n", nDevices);
    }
    for (i = 0; i < nDevices; i++) {
        checkCudaErrors(hipGetDeviceProperties(&prop, i), "hipGetDeviceProperties", __FILE__, __LINE__);
        printf("\nDevice %d/%d: \"%s\"\n", i+1, nDevices, prop.name);
        printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("  Memory Clock Rate (GHz): %f\n", prop.memoryClockRate/1.0e6);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Global Memory (GB): %f\n", prop.totalGlobalMem/8.0e9);
        printf("  Constant Memory (Bytes): %zu\n", prop.totalConstMem);
        printf("  Max mem pitch: %ld\n", prop.memPitch);
        printf("  Texture Alignment: %ld\n", prop.textureAlignment);
        printf("  Warp Size : %d\n", prop.warpSize);
        printf("  Max Threads per Block: %d\n", prop.maxThreadsPerBlock);
        printf("  Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("  Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
        printf("  Registers per mp: %d\n\n", prop.regsPerBlock);
    }


    fptr = fopen("param_2d.txt", "r");
    if(fptr != NULL) {

        if(fgets(readString, 100, fptr) != NULL) {
            n = atoi(readString);
        } else {
            printf("Not able to read n\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            nxBlocks = atoi(readString);
        } else {
            printf("Not able to read nxBlocks\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            nyBlocks = atoi(readString);
        } else {
            printf("Not able to read nyBlocks\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            blockxSize = atoi(readString);
        } else {
            printf("Not able to read blockxSize\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            blockySize = atoi(readString);
        } else {
            printf("Not able to read blockySize\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            it_max = atoi(readString);
        } else {
            printf("Not able to read it_max\n");
        }

        if(fgets(readString, 100, fptr) != NULL) {
            it_print = atoi(readString);
        } else {
            printf("Not able to read it_print\n");
        }

    } else {
        printf("Not able to open the file param.txt\n");
    }
    fclose(fptr);

    L = 1.0;
    h = L / (double) (n-1);

    printf("nb on grid points = %d x %d\n", n, n);
    printf("dim of grid = %.1f x %.1f\n", L, L);
    printf("step = %f\n\n", h);


    nWorkers_x = nxBlocks * blockxSize;
    printf("nb of blocks (x) = %d\n", nxBlocks);
    printf("size of blocks (x) = %d\n", blockxSize);
    printf("nb of workers (x) = %d\n\n", nWorkers_x);
    if(nWorkers_x > n) {
        printf("increase n, or decrease nxBlocks and/or blockxSize");
        exit(0);
    }

    nWorkers_y = nyBlocks * blockySize;
    printf("nb of blocks (y) = %d\n", nyBlocks);
    printf("size of blocks (y) = %d\n", blockySize);
    printf("nb of workers (y) = %d\n\n", nWorkers_y);
    if(nWorkers_y > n) {
        printf("increase n, or decrease nyBlocks and/or blockySize");
        exit(0);
    }


    ntx = n;
    nty = n;
    ntx_local = n / nWorkers_x;
    nty_local = n / nWorkers_y;

    printf("ntx = %d\n", ntx);
    printf("nty = %d\n", nty);
    printf("ntx_local = %d\n", ntx_local);
    printf("nty_local = %d\n\n", nty_local);
    if(ntx != ntx_local*nWorkers_x) {
        printf("bad set of parameters (x) !");
        exit(0);
    }
    if(nty != nty_local*nWorkers_y) {
        printf("bad set of parameters (y) !");
        exit(0);
    }


    size_u = ntx * nty * sizeof(double);
    printf("Size of d_u = %.2f MB \n\n", (double)size_u/(1024.0*1024.0));

    size_err = nyBlocks * sizeof(double);

    checkCudaErrors(hipMalloc(&d_u, size_u), "hipMalloc", __FILE__, __LINE__);

    checkCudaErrors(hipMalloc(&d_unew, size_u), "hipMalloc", __FILE__, __LINE__);

    checkCudaErrors(hipMalloc(&d_err, size_err), "hipMalloc", __FILE__, __LINE__);



    h_err = (double*) malloc(size_err);
    if(h_err == NULL) {
        fprintf(stderr, "Memory allocation failed for h_err\n");
        exit(0);
    }


    dim3 dimGrid(nxBlocks, nyBlocks, 1);
    dim3 dimBlock(blockxSize, blockySize, 1);


    init_2d_kernel<<<dimGrid, dimBlock>>>(ntx, ntx_local, nty_local, nWorkers_x, nWorkers_y, d_u);
    checkCudaErrors(hipGetLastError(), "hipGetLastError", __FILE__, __LINE__);
    checkCudaErrors(hipDeviceSynchronize(), "hipDeviceSynchronize", __FILE__, __LINE__);


    it = 1;
    while(it <= it_max) {

        if(it%2 != 0) {
            compute_2d_kernel<<<dimGrid, dimBlock>>>(ntx, nty, ntx_local, nty_local, nWorkers_x, nWorkers_y, h, d_u, d_unew);
            checkCudaErrors(hipGetLastError(), "hipGetLastError", __FILE__, __LINE__);
            checkCudaErrors(hipDeviceSynchronize(), "hipDeviceSynchronize", __FILE__, __LINE__);
        } else {
            compute_2d_kernel<<<dimGrid, dimBlock>>>(ntx, nty, ntx_local, nty_local, nWorkers_x, nWorkers_y, h, d_unew, d_u);
            checkCudaErrors(hipGetLastError(), "hipGetLastError", __FILE__, __LINE__);
            checkCudaErrors(hipDeviceSynchronize(), "hipDeviceSynchronize", __FILE__, __LINE__);
        }

        if(it%it_print == 0) {
            max_error_kernel<<<nyBlocks, blockySize, size_err>>>(ntx, nty, nty_local, nWorkers_y, h, d_u, d_err);
            checkCudaErrors(hipDeviceSynchronize(), "hipDeviceSynchronize", __FILE__, __LINE__);
            checkCudaErrors(hipGetLastError(), "hipGetLastError", __FILE__, __LINE__);
            checkCudaErrors(hipMemcpy(h_err, d_err, size_err, hipMemcpyDeviceToHost), "hipMemcpy", __FILE__, __LINE__);
            err = h_err[0];
            for (i = 1; i < nyBlocks; i++) {
                if(err < h_err[i]) {
                    err = h_err[i];
                }
            }
            printf("it = %d/%d, error = %f\n", it, it_max, err);
        }

        it++;
    }

    free(h_err);


    checkCudaErrors(hipFree(d_u), "hipFree", __FILE__, __LINE__);
    checkCudaErrors(hipFree(d_unew), "hipFree", __FILE__, __LINE__);
    checkCudaErrors(hipFree(d_err), "hipFree", __FILE__, __LINE__);

    checkCudaErrors(hipEventRecord(stop), "hipEventRecord", __FILE__, __LINE__);
    checkCudaErrors(hipEventSynchronize(stop), "hipEventSynchronize", __FILE__, __LINE__);
    checkCudaErrors(hipEventElapsedTime(&tt, start, stop), "hipEventElapsedTime", __FILE__, __LINE__);

    printf("Ellapsed time = %.3f sec", tt / 1000.0f);

    return 0;
}

