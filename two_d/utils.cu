

#include <hip/hip_runtime.h>
#include <iostream>

const int threadsPerBlock = 256;


void checkCudaErrors(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}



__global__ void max_error(int ntx, int nty, int nty_local, int n_Workers, double h, double *u, double *err) {

    //__shared__ double cache[threadsPerBlock];
    extern __shared__ double cache[];

    int tid;
    int i, cacheIndex;
    int l, ll;
    int j, jj0, jj1, jy;

    double x, y;
    double tmp, err_local;


    tid = threadIdx.x + blockIdx.x * blockDim.x;
    cacheIndex = threadIdx.x;
    err_local = 0.0;

    while (tid < n_Workers) {

        jj0 = nty_local * tid;
        jy = jj0 - 2 * tid - 1;

        for(j = 1; j < nty_local-1; j++) {

            y = (double) (jy + j) * h;

            jj1 = (jj0 + j) * ntx;

            for(l = 1; l < ntx-1; l++) {

                x = (double) l * h;

                ll = jj1 + l;

                tmp = fabs(u[ll] - x * y * (x - 1.0) * (y - 1.0));
                if(tmp > err_local) {
                    err_local = tmp;
                }
            }
        }

        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = err_local;
    __syncthreads();

    i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            err_local = cache[cacheIndex];
            tmp = cache[cacheIndex + i];
            if(tmp > err_local) {
                cache[cacheIndex] = tmp;
            }
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        err[blockIdx.x] = cache[0];
    }

}


