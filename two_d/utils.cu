

#include <hip/hip_runtime.h>
#include <iostream>


void checkCudaErrors(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

